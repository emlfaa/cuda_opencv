#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "cuda_fuction.h"

#define THREAD_DIM_X 512

inline __device__ __host__ float regin_union(float a1, float a2, float b1, float b2) {
    return (min)(a2, b2) - (max)(a1, b1);
}

// Down resize
__global__ void _resize_kernel(uint8_t *src_pix,
                               uint8_t *dst_pix, 
                               int srcWidth, 
                               int srcHeight,
                               int srcChannel,
                               int dstWidth,
                               int dstHeight
                               )
{
    const int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dc = gidx % srcChannel;
    const int dx = gidx / srcChannel % dstWidth;
    const int dy = gidx / srcChannel / dstWidth;
    
    if(dy >= dstHeight) return;

    const float bw = (float) srcWidth / (float) dstWidth;
    const float bh = (float) srcHeight / (float) dstHeight;
    const float fx1 = dx * bw;
    const float fx2 = (dx + 1) * bw;
    const float fy1 = dy * bh;
    const float fy2 = (dy + 1) * bh;
    const int st_x = (int) floor(fx1), ed_x = ceil(fx2);
    const int st_y = (int) floor(fy1), ed_y = ceil(fy2);
    float sv = 0;
    for(int x = st_x; x < ed_x; x++) {
        float px = regin_union(x, x + 1.0f, fx1, fx2);
        for(int y = st_y; y < ed_y; y++) {
            float py = region_union(y, y + 1.0f, fy1, fy2);
            sv += src_pix[srcChannel * (x + srcWidth * y) + dc] * px * py;
        }
    }

    sv /= bw * bh;
    dst_pix[srcChannel * (dy * dstHeight + dx) + dc] = (uint8_t) sv;
}

// RGB to gray
__global__ void bgr2gray_kernel(const uchar3 *src, 
                                const int srcWidth,
                                const int srcHeight, 
                                unsigned char *dst) {
    
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if(idx < srcWidth && idy < srcHeight)
    {
        uchar3 rgb = src[idy * srcWidth + idx];
        dst[idy * srcWidth + idx] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
    }
}

void cuda_function::ImgDownSampling() {

    uint8_t *img_gpu;
    uint8_t *imgSmall_gpu;

    // cuda 空间申请
    hipMalloc((void**) &img_gpu, ImageSize);
    hipMalloc((void**) &imgSmall_gpu, imageSize / scale / scale);

    hipMemcpy(img_gpu, InputImage.data, imageSize, hipMemcpyHostToDevice);

    if(img_gpu == NULL || imgSmall_gpu == NULL)
    {
        printf("ImgDownSampling() create cuda memory is failed！！！！！！");
        std::exit(0);
    }

    const int SW = ImageWidth / scale;
    const int SH = ImageHeight / scale;

    _resize_kernel<<<(SW * 3 * SH + THREAD_DIM_X - 1) / THREAD_DIM_X, THREAD_DIM_X>>>(img_gpu, imgW, imgH, 3, imgSmall_gpu, SW, SH);
    img_small.create(imgH / scale, imgW / scale, CV_8UC3);
    hipMemcpy(img_small.data, imgSmall_gpu, SW * 3 * SH, hipMemcpyDeviceToHost);
}

void cuda_function::ImgRGB2GRAY() {

    uchar3 *img_input;
    unsigned char *img_output; 

    hipMalloc((void**)&img_input, ImageWidth*ImageHeight*sizeof(uchar3));
    hipMalloc((void**)&img_output, ImageWidth * ImageHeight * sizeof(unsigned char));
    hipMemcpy(img_input, InputImage.data, ImageWidth*ImageHeight*sizeof(uchar3), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((ImageWidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ImageHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    _rgb2gray<<<blocksPerGrid, threadsPerBlock>>>(img_input, image)
}